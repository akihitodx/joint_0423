#include "hip/hip_runtime.h"
#include "Graph.cuh"
#include "fstream"
#include "iostream"
#include "set"
#include "queue"
#include "unordered_map"
#include "unordered_set"
#include <iomanip>

using namespace std;

struct CompareFirst {
    bool operator()(const std::pair<int, int>& lhs, const std::pair<int, int>& rhs) const {
        return lhs.first < rhs.first;
    }
};
bool Graph::ReadInFile(const std::string &FileName, int flag) {
    char type;
    int vv_num, ee_num;
    int a,b,c;
    ifstream infile;
    cout<<"Read File: "<<FileName <<endl;
    infile.open(FileName, ios::in);
    if (!infile.is_open()) {
        cout << "file open failed" << endl;
        return false;
    }
    infile >> type >> vv_num >> ee_num;
    v_num = vv_num;
    e_num = ee_num;
    q_h_node.resize(v_num,0);
    q_h_label.resize(v_num,-1);
    q_h_degree.resize(v_num,-1);
    originalId.resize(v_num, -1);
    vector<vector<int>> adj_temp(v_num);
    vector<multiset<pair<int,int>,CompareFirst>> adj_L_temp(v_num);

    for(int i = 0 ; i <v_num; ++i){
        infile >>type >> a>> b>> c;
        originalId[i] = a;
        q_h_label[a] = b;
        q_h_degree[a] = c;
    }
    for(int i = 0; i <e_num; ++i){
        infile >>type >> a>> b>> c;
//        adj_temp[a].insert(b);
//        adj_temp[b].insert(a);
        adj_L_temp[a].insert({q_h_label[b],b});
        adj_L_temp[b].insert({q_h_label[a],a});
    }
    for(int i = 0; i<v_num; ++i){
        for(auto ele : adj_L_temp[i]){
            adj_temp[i].push_back(ele.second);
        }
    }
    q_h_adj.clear();
    q_h_adj.insert(q_h_adj.end(),adj_temp[0].begin(), adj_temp[0].end());
    int count = 0;
    for(int i = 1; i<q_h_node.size(); ++i){
        count += (int)adj_temp[i-1].size();
        q_h_node[i] = count;
        q_h_adj.insert(q_h_adj.end(),adj_temp[i].begin(), adj_temp[i].end());
    }
    if(flag == 1) {
        adj_vs.resize(adj_temp.size());
        for(int i = 0 ; i< v_num ;++i){
            adj_vs[i].insert(adj_temp[i].begin(), adj_temp[i].end());
        }
    }
    cutStep = 0;
    midNode = vector<vector<int>>(q_h_label.size(), vector<int>());
    joint_group = vector<vector<Tag4>>(q_h_label.size(), vector<Tag4>());
    cout << "read finish" << endl;
    print();
    return true;
}

void Graph::except_ring() {
    auto adj_temp = adj_vs;
    auto adj_update = adj_vs;
    queue<int> queue;
    unordered_set<int> used;
    queue.push(0);
    while(!queue.empty()){
        int id = queue.front();
        queue.pop();
        used.insert(id);
        for(auto i : adj_temp[id]){
            if (used.count(i) > 0){
                single_edge.push_back(i);
                single_edge.push_back(id);
                adj_update[i].erase(id);
                adj_update[id].erase(i);
                adj_temp[i].erase(id);
            }else{
                queue.push(i);
                used.insert(i);
                adj_temp[i].erase(id);
            }
        }
    }

    vector<multiset<pair<int,int>,CompareFirst>> adj_L_temp(v_num);
    for(int i = 0 ;i < v_num; ++i){
        for(auto ele : adj_update[i]){
            adj_L_temp[i].insert({q_h_label[ele],ele});
        }
    }
    int count = 0;
    for(int i = 0; i < v_num; ++i){
        q_h_degree[i] = (int)adj_update[i].size();
        count += (int)adj_update[i].size();
        if(i+1 < q_h_node.size()) q_h_node[i+1] = count;
    }
    q_h_adj.clear();
    for(auto &i : adj_L_temp){
        for(auto ele : i){
            q_h_adj.push_back(ele.second);
        }
    }
    cout<<"finish except ring"<<endl;
    for(auto i : this->single_edge){
        cout<<i <<" ";
    }
    cout<<endl;
    print();
}

void GuangDu(vector<int>& q_h_level, vector<int>& q_h_node, vector<int>& q_h_adj, int level, int vId) {
    int next;
    if (vId < q_h_node.size() - 1) {
        next = q_h_node[vId + 1];
    }
    else {
        next = q_h_adj.size();
    }
    bool isAllNotFu1 = true;
    for (int i = q_h_node[vId]; i < next; i++) {
        if (q_h_level[q_h_adj[i]] == -1) {
            q_h_level[q_h_adj[i]] = level;
            isAllNotFu1 = false;
        }
    }
    if (isAllNotFu1) {
        return;
    }
    else {
        for (int i = q_h_node[vId]; i < next; i++) {
            GuangDu(q_h_level, q_h_node, q_h_adj, level + 1, q_h_adj[i]);
        }
    }
}

bool Graph::calcLevelId() {
    //找到度最小的节点
    int minDu = q_h_node.size();
    for (int i = 0; i < q_h_node.size(); i++) {
        if (minDu > q_h_degree[i]) {
            minDu = q_h_degree[i];
            minLevelId = i;
        }
    }
    vector<int> q_h_level(q_h_node.size(), -1);
    q_h_level[minLevelId] = 0;
    GuangDu(q_h_level, q_h_node, q_h_adj, 1, minLevelId);
    int max = 0;
    for (int i = 0; i <= q_h_level.size() - 1; i++) {
        if (max < q_h_level[i]) {
            max = q_h_level[i];
            maxLevelId = i;
        }
    }
    return true;
}

 bool Graph::division(int count, int & name) {
    cout << "start split" << endl;
//    print();

    q_h_adj_chai.resize(q_h_adj.size(), -1);
    queue<int> leftQueue;
    queue<int> rightQueue;
    //左右队列放入元素
    leftQueue.push(maxLevelId);
    rightQueue.push(minLevelId);
    while (!(leftQueue.empty() && rightQueue.empty())) {
        int checkV = -1;
        int searchV = -1;
        while (!leftQueue.empty() && checkV == -1) {
            //从left队列取出第一个
            searchV = leftQueue.front();
            //判断有没有可取点,若有选取一个点
            for (int i = q_h_node[searchV]; i < q_h_node[searchV] + q_h_degree[searchV]; i++) {
                if (q_h_adj_chai[i] == -1) {
                    //找到可取点，设置checkV为选中点id，将this_chai的这个位置设置为当前编号time
                    checkV = q_h_adj[i];
                    leftQueue.push(checkV);
                    q_h_adj_chai[i] = 0;
                    break;
                }
            }
            //没有可取点,队列弹出
            if (checkV == -1) {
                leftQueue.pop();
            }
        }
        //找到另一个边
        if (checkV != -1 && searchV != -1) {
            for (int i = q_h_node[checkV]; i < q_h_node[checkV] + q_h_degree[checkV]; i++) {
                if (q_h_adj[i] == searchV) {
                    q_h_adj_chai[i] = 0;
                    break;
                }
            }
        }
        checkV = -1;
        searchV = -1;
        while (!rightQueue.empty() && checkV == -1) {
            //从left队列取出第一个
            searchV = rightQueue.front();
            //判断有没有可取点,若有选取一个点
            for (int i = q_h_node[searchV]; i < q_h_node[searchV] + q_h_degree[searchV]; i++) {
                if (q_h_adj_chai[i] == -1) {
                    //找到可取点，设置checkV为选中点id，将this_chai的这个位置设置为当前编号time
                    checkV = q_h_adj[i];
                    rightQueue.push(checkV);
                    q_h_adj_chai[i] = 1;
                    break;
                }
            }
            //没有可取点,队列弹出
            if (checkV == -1) {
                rightQueue.pop();
            }
        }
        //找到另一个边
        if (checkV != -1 && searchV != -1) {
            for (int i = q_h_node[checkV]; i < q_h_node[checkV] + q_h_degree[checkV]; i++) {
                if (q_h_adj[i] == searchV) {
                    q_h_adj_chai[i] = 1;
                    break;
                }
            }
        }
    }
    //扫描分割点

    int get = -1;

    for (int i = 0; i < q_h_node.size(); i++) {
        for (int j = q_h_node[i]; j < q_h_node[i] + q_h_degree[i] - 1; j++) {
            if (q_h_adj_chai[j] != q_h_adj_chai[j + 1]) {
                get = originalId[i];
                break;
            }
        }
    }

    cout << "split finish" << endl;
    print();

    vector<int> leftQMap;//偏移量数组
    vector<int> rightQMap;
    vector<int> leftLabelMap;//标签数组
    vector<int> rightLabelMap;
    vector<int> leftAdjMap;//临接点数组
    vector<int> rightAdjMap;

    vector<int> leftOldQMap;
    vector<int> rightOldQMap;
    vector<int> leftOldQMapOri;
    vector<int> rightOldQMapOri;
    vector<int> leftDuMap;//上一个点id
    vector<int> rightDuMap;
    //构建左右两个GSI
    for (int i = 0; i < q_h_node.size(); i++) {
        bool isLeftPush = false;
        bool isRightPush = false;
        int thisLeftQ = leftAdjMap.size();
        int thisRightQ = rightAdjMap.size();
        for (int j = q_h_node[i]; j < q_h_node[i] + q_h_degree[i]; j++) {
            if (q_h_adj_chai[j] == 0) {
                leftAdjMap.push_back(q_h_adj[j]);
                isLeftPush = true;
            }
            if (q_h_adj_chai[j] == 1) {
                rightAdjMap.push_back(q_h_adj[j]);
                isRightPush = true;
            }
        }
        if (isLeftPush) {
            leftQMap.push_back(thisLeftQ);
            leftOldQMap.push_back(i);
            leftOldQMapOri.push_back(originalId[i]);
            leftLabelMap.push_back(q_h_label[i]);
        }
        if (isRightPush) {
            rightQMap.push_back(thisRightQ);
            rightOldQMap.push_back(i);
            rightOldQMapOri.push_back(originalId[i]);
            rightLabelMap.push_back(q_h_label[i]);
        }
    }
    //邻接点id转为新图id
    for (int i = 0; i < leftAdjMap.size(); i++) {
        for (int j = 0; j < leftOldQMap.size(); j++) {
            if (leftAdjMap[i] == leftOldQMap[j]) {
                leftAdjMap[i] = j;
                break;
            }
        }
    }
    for (int i = 0; i < rightAdjMap.size(); i++) {
        for (int j = 0; j < rightOldQMap.size(); j++) {
            if (rightAdjMap[i] == rightOldQMap[j]) {
                rightAdjMap[i] = j;
                break;
            }
        }
    }
    //设置新图的度数组
    for (int i = 0; i < leftQMap.size(); i++) {
        if (i < leftQMap.size() - 1) {
            leftDuMap.push_back(leftQMap[i + 1] - leftQMap[i]);
        }
        else {
            leftDuMap.push_back(leftAdjMap.size() - leftQMap[i]);
        }
    }
    for (int i = 0; i < rightQMap.size(); i++) {
        if (i < rightQMap.size() - 1) {
            rightDuMap.push_back(rightQMap[i + 1] - rightQMap[i]);
        }
        else {
            rightDuMap.push_back(rightAdjMap.size() - rightQMap[i]);
        }
    }

    //设置两个分割后子图的属性
    Graph left = Graph();
    Graph right = Graph();
    left.q_h_node.assign(leftQMap.begin(), leftQMap.end());
    left.q_h_label.assign(leftLabelMap.begin(), leftLabelMap.end());
    left.q_h_degree.assign(leftDuMap.begin(), leftDuMap.end());
    left.q_h_adj.assign(leftAdjMap.begin(), leftAdjMap.end());
    left.originalId.assign(leftOldQMapOri.begin(), leftOldQMapOri.end());
    this->leftChild = &left;
    left.father = this;

    left.group_name = ++name;

    right.q_h_node.assign(rightQMap.begin(), rightQMap.end());
    right.q_h_label.assign(rightLabelMap.begin(), rightLabelMap.end());
    right.q_h_degree.assign(rightDuMap.begin(), rightDuMap.end());
    right.q_h_adj.assign(rightAdjMap.begin(), rightAdjMap.end());
    right.originalId.assign(rightOldQMapOri.begin(), rightOldQMapOri.end());
    this->rightChild = &right;
    right.father = this;

    right.group_name = ++name;
    cout << "split over" << endl;
    cout << "left subgraph" << endl;


    group_name_map[left.group_name].insert(left.originalId.begin(), left.originalId.end());
    group_name_map[right.group_name].insert(right.originalId.begin(), right.originalId.end());

    left.print();
    cout << "right subgraph" << endl;
    right.print();

    joint_group[count].emplace_back(Tag4({left.group_name,right.group_name,get, this->group_name}));
    //如果拆分后的子图节点的个数大于2的话，递归进行下一次分割
    if (left.q_h_node.size() > 2) {
        left.calcLevelId();
        left.division(count + 1,name);
    }
    else {

        int another = 0;
        if(left.originalId[0]==get){
            another = left.originalId[1];
        }else{
            another = left.originalId[0];
        }
        single_pair.push_back(get);
        single_pair.push_back(another);
        single_pair_name.push_back(left.group_name);
    }

    if (right.q_h_node.size() > 2) {
        right.calcLevelId();
        right.division(count + 1,name);
    }
    else {
        int another = 0;
        if(right.originalId[0]==get){
            another = right.originalId[1];
        }else{
            another = right.originalId[0];
        }
        single_pair.push_back(get);
        single_pair.push_back(another);
        single_pair_name.push_back(right.group_name);
    }
    if (cutStep < count) {
        cutStep = count;
    }
    return true;

}

void Graph::print() {
    cout << "=============================================" << endl;
    cout << setw(12) << "node:";

    for (int i = 0; i < q_h_node.size(); i++)
        cout << setw(3) << q_h_node[i];
    cout << endl;
    cout << setw(12) << "label:";
    for (int i = 0; i < q_h_label.size(); i++)
        cout << setw(3) << q_h_label[i];
    cout << endl;
    cout << setw(12) << "originalId:";
    for (int i = 0; i < originalId.size(); i++)
        cout << setw(3) << originalId[i];
    cout << endl;
    cout << setw(12) << "degree:";
    for (int i = 0; i < q_h_degree.size(); i++)
        cout << setw(3) << q_h_degree[i];
    cout << endl;
    cout << setw(12) << "adj:";
    for (int i = 0; i < q_h_adj.size(); i++)
        cout << setw(3) << q_h_adj[i];
    cout << endl;
    cout << "=============================================" << endl;
}

