#include "hip/hip_runtime.h"
#include "alg.cuh"
#include "iostream"
__device__ bool add_tag(Tag5 tag,Tag5* index,Tag5* row_res,int tid_data,int data_size, size_t pitch,int next){
    if(next == 0){
        int it = atomicAdd(&row_res[0].data[1], 1) + 1;
        if(it <= row_res[0].data[2]){
            printf("insert self %d---%d,%d\n",row_res[0].data[1],row_res[0].data[2],tid_data);
            row_res[it].data[0] = tag.data[0];
            row_res[it].data[1] = tag.data[1];
            row_res[it].data[2] = tag.data[2];
            row_res[it].data[3] = tag.data[3];
            row_res[it].data[4] = tag.data[4];
            return true;
        }else{
            return false;
        }
    }else{
        printf("try insert %d -to- >%d\n",tid_data,(tid_data + next)%data_size);
        Tag5* row_next =  (Tag5*)((char*)index + pitch * ((tid_data + next)%data_size));
        if(row_next[0].data[3] == tid_data){
            printf("%d == data[3]\n",tid_data);
            int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
            if(new_size >= row_next[0].data[1]){
                printf("insert others %d---%d,%d--to--> %d\n",row_next[0].data[1],row_next[0].data[2],tid_data,(tid_data + next)%data_size);
                int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
                row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
                row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
                row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
                row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
                row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
                return true;
            }else{
                printf("others failed %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
                atomicAdd(&row_next[0].data[2], 1);
                return false;
            }
        }else if(row_next[0].data[3] == -1 && row_next[0].data[2] > row_next[0].data[1]){
            printf("%d == -1 >data[1]\n",tid_data);
            int old = atomicAdd(&row_next[0].data[3],tid_data+1);
            if(old != -1){
                printf("%d --> %d update failed\n",tid_data,(tid_data + next)%data_size);
                atomicSub(&row_next[0].data[3],tid_data+1);
                return false;
            }
            else{
                int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
                if(new_size >= row_next[0].data[1]){
                    printf("%d --> %d update succeed and insert\n",tid_data,(tid_data + next)%data_size);
                    printf("%d---%d,%d\n",row_next[0].data[1],row_next[0].data[2],tid_data);
                    int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
                    row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
                    row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
                    row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
                    row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
                    row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
                    return true;
                }else{
                    printf("others failed but in %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
                    atomicAdd(&row_next[0].data[2], 1);
                    return false;
                }
            }
        }else{
            printf("%d shit!\n",tid_data);
            printf("over and next %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
            return false;
        }
    }
}


//__device__ bool add_tag_old(Tag5 tag,Tag5* index,Tag5* row_res,int tid_data,int data_size, size_t pitch,int next){
//    if(next==0){
//        int it = atomicAdd(&row_res[0].data[1], 1) + 1;
//        if(it <= row_res[0].data[2]){
//            printf("insert self %d---%d,%d\n",row_res[0].data[1],row_res[0].data[2],tid_data);
//            row_res[it].data[0] = tag.data[0];
//            row_res[it].data[1] = tag.data[1];
//            row_res[it].data[2] = tag.data[2];
//            row_res[it].data[3] = tag.data[3];
//            row_res[it].data[4] = tag.data[4];
//        }else{
//            printf("need insert others %d\n",tid_data);
//            ++next;
//            add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//        }
//    }else{
//        printf("try insert %d -to- >%d\n",tid_data,(tid_data + next)%data_size);
//        Tag5* row_next =  (Tag5*)((char*)index + pitch * ((tid_data + next)%data_size));
//        printf("%d --> %d : %d , %d ,%d ,%d\n",tid_data,(tid_data + next)%data_size,row_next[0].data[1],row_next[0].data[2],row_next[0].data[3],row_next[0].data[4]);
//        if(row_next[0].data[3] == tid_data){
//            printf("%d == data[3]\n",tid_data);
//            int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
//            if(new_size >= row_next[0].data[1]){
//                printf("insert others %d---%d,%d--to--> %d\n",row_next[0].data[1],row_next[0].data[2],tid_data,(tid_data + next)%data_size);
//                int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
//                row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
//                row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
//                row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
//                row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
//                row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
//            }else{
//                printf("others failed %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
//                atomicAdd(&row_next[0].data[2], 1);
//                ++next;
//                add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//            }
//        }else if(row_next[0].data[3] == -1 && row_next[0].data[2] > row_next[0].data[1]){
//            printf("%d == -1 >data[1]\n",tid_data);
//            int old = atomicAdd(&row_next[0].data[3],tid_data+1);
//            if(old != -1){
//                printf("%d --> %d update failed\n",tid_data,(tid_data + next)%data_size);
//                atomicSub(&row_next[0].data[3],tid_data+1);
//                ++next;
//                add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//            }
//            else{
//
//                int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
//                if(new_size >= row_next[0].data[1]){
//                    printf("%d --> %d update succeed and insert\n",tid_data,(tid_data + next)%data_size);
//                    printf("%d---%d,%d\n",row_next[0].data[1],row_next[0].data[2],tid_data);
//                    int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
//                    row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
//                    row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
//                    row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
//                    row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
//                    row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
//                }else{
//                    printf("others failed but in %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
//                    atomicAdd(&row_next[0].data[2], 1);
//                    ++next;
//                    add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//                }
//            }
//        }else{
//            printf("%d shit!\n",tid_data);
//            printf("over and next %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
//            ++next;
//            add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//        }
//
//    }
//    return true;
//}


__global__ void data_filter(Tag5* index,
                                size_t pitch,
                                const int* data_node,
                                const int* data_degree,
                                const int* data_label,
                                const int* data_adj,
                                const int* query_node,
                                const int* query_degree,
                                const int* query_label,
                                const int* query_adj,
                                int data_size,
                                int query_size){
    int tid_data = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_query = blockIdx.y * blockDim.y + threadIdx.y;
    if(tid_data< data_size && tid_query< query_size){
        if(query_label[tid_query] != data_label[tid_data]) return;
        if(query_degree[tid_query] > data_degree[tid_data]) return;
        int query_len = query_node[tid_query] + query_degree[tid_query];
        int data_len = data_node[tid_data] + data_degree[tid_data];
        int query_loc = query_node[tid_query], data_loc = data_node[tid_data];
//        printf("%d %d\n",tid_data,tid_query);
        while(query_loc < query_len && data_loc < data_len){
//            printf("%d-%d, %d------%d, %d^^^%d\n",tid_data,tid_query,data_label[data_adj[data_loc]],query_label[query_adj[query_loc]],data_loc,query_loc);
            if(query_label[query_adj[query_loc]] == data_label[data_adj[data_loc]]){
                ++query_loc;
                ++data_loc;
            }else if(query_label[query_adj[query_loc]] < data_label[data_adj[data_loc]]){
                break;
            }else{
                ++data_loc;
            }
        }
        if(query_loc == query_len){
//            printf("-----%d %d\n",tid_data,tid_query);
            Tag5* row_res = (Tag5*)((char*)index + pitch * tid_data);
            Tag5 tag = {0,tid_data,0,0,tid_query};
            int next = 0;
            bool flag = false;
            while (!flag && next<data_size){
                flag = add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
                ++next;
            }

        }
    }
}


//    test
__host__ void print_h_index(Tag5* h_index,int data_size){
    cout << "=========" << endl;
    for(int i = 0; i<data_size; ++i){
        cout<<i<<": ";
        for(int j = 0 ; j<h_index[i*N_size].data[0]; ++j){
            cout<<h_index[i*N_size + j].data[0]<<" "
                <<h_index[i*N_size + j].data[1]<<" "
                <<h_index[i*N_size + j].data[2]<<" "
                <<h_index[i*N_size + j].data[3]<<" "
                <<h_index[i*N_size + j].data[4]<<" / ";
        }
        cout<<endl;
    }
}

__global__ void d_print(Tag5* index,size_t pitch,int size,int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        for(int i = 0 ; i< N ;++i){
            printf("%d: %d %d %d %d %d\n",tid,row[i].data[0],row[i].data[1],row[i].data[2],row[i].data[3],row[i].data[4]);
        }

    }
}

__global__ void add_one(Tag5* index,size_t pitch,Tag5 tag, int loc,int data_v_num){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if( tid == 11 || tid == 10 || tid == 0 || tid == 1 || tid == 2){
        printf("tid --> %d\n",tid);
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        int next = 0;
        bool flag = false;
        while (!flag && next<data_v_num){
            flag = add_tag(tag,index,row,tid,data_v_num,pitch,next);
            ++next;
        }
    }
}

__global__ void init_edge(Tag5* index,size_t pitch,
                          const int* query_edge_pair,
                          const int* data_label,
                          const int* data_node,
                          const int* data_degree,
                          const int* data_adj,
                          const int* single_group_name,
                          int data_v_num, int edge_num){
    int tid_data = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_query = blockIdx.y * blockDim.y + threadIdx.y;
    if(tid_data< data_v_num && tid_query< edge_num){
        int node_first = query_edge_pair[2* tid_query];
        int node_second = query_edge_pair[2* tid_query + 1];
        bool flag = false;
        Tag5 *row = (Tag5*)((char*)index + pitch * tid_data);
        for(int i = 1 ; i <= row[0].data[1]; ++i){
            if(row[i].data[0] == 0 && row[i].data[4] == node_first){
                flag = true;
                break;
            }
        }
        if(flag){
//            printf("%d--%d == %d %d\n",tid_data,tid_query, node_first, node_second);
            for(int node = data_node[tid_data]; node <data_node[tid_data] + data_degree[tid_data]; ++node){
//                printf("%d, %d ---%d\n",tid_data,node,data_node[tid_data] + data_degree[tid_data]);
                int adj_node = data_adj[node];
//                printf("%d--%d == %d %d ---%d\n",tid_data,tid_query, node_first, node_second ,adj_node);
                Tag5 *row_adj = (Tag5*)((char*)index + pitch * adj_node);
                for(int i = 1 ; i<= row_adj[0].data[1]; ++i){
                    if(row_adj[i].data[0] == 0 && row_adj[i].data[4] == node_second){
                        Tag5 tag_first = {single_group_name[tid_query],tid_data,i-1,adj_node,node_first};
                        Tag5 tag_second = {single_group_name[tid_query],tid_data,i-1,tid_data,node_second};
                        printf("add tag %d, %d\n",tid_data,adj_node);
                        int next = 0;
                        bool ff = false;
                        while(!ff && next<data_v_num){
                            ff = add_tag(tag_first,index,row,tid_data,data_v_num,pitch,next);
                            ++next;
                        }
                        next = 0;
                        ff = false;
                        while(!ff && next<data_v_num){
                            ff = add_tag(tag_second,index,row_adj,adj_node,data_v_num,pitch,next);
                            ++next;
                        }
                        break;
                    }
                }
            }
        }
    }
}

__device__ int find_next(int cur_tid,int ori_tid,Tag5* index,size_t pitch){
    ++cur_tid;
    Tag5 *row = (Tag5*)((char*)index + pitch * cur_tid);
    while (row[0].data[3] != ori_tid){
        ++cur_tid;
        row = (Tag5*)((char*)index + pitch * cur_tid);
    }
    return cur_tid;
};

__global__ void joint(Tag5* index,size_t pitch,Tag3 info,
                      const int* del_edge,
                      const int* node_set,
                      const int* degree_set,
                      const int* adj_set,
                      int data_v_num){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 10 || tid == 11){
        Tag2 first_set[N_size/2];
        Tag2 second_set[N_size/2];
        int first_count = 0, second_count = 0;
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        for (int i = 1 ; i <= row[0].data[2]; ++i){
            if(row[i].data[0] == info.data[0] && row[i].data[4] == info.data[2]){
                first_set[first_count++] = {tid,i};
                break;
            }
            if(row[i].data[0] == info.data[1] && row[i].data[4] == info.data[2]){
                second_set[second_count++] = {tid,i};
                break;
            }
        }
        if(row[0].data[1] > row[0].data[2]){
            int cur_count = row[0].data[2];
            int sum_count = row[0].data[1];
            int cur_tid = tid;
            int next_tid = find_next(cur_tid,tid,index,pitch);
            while (cur_count < sum_count){
                Tag5 *row_next = (Tag5*)((char*)index + pitch * next_tid);
                cur_count += row_next[0].data[4];
                for (int i = 0 ; i < row_next[0].data[4]; ++i){
                    if(row_next[i].data[0] == info.data[0] && row_next[i].data[4] == info.data[2]){
                        first_set[first_count++] = {next_tid,N_size- i -1};
                        break;
                    }
                    if(row_next[i].data[0] == info.data[1] && row_next[i].data[4] == info.data[2]){
                        second_set[second_count++] = {next_tid,N_size- i -1};
                        break;
                    }
                }
            }
        }
        for(int i = 0 ;i < second_count; ++i){
            printf("%d,%d\n",second_set[i].data[0],second_set[i].data[1]);
        }
    }

}