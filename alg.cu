#include "hip/hip_runtime.h"
#include "alg.cuh"
#include "iostream"

__device__ bool add_tag(Tag5 tag,Tag5* index,Tag5* row_res,int tid_data,int data_size, size_t pitch,int next){
    if(next==0){
        int it = atomicAdd(&row_res[0].data[1], 1) + 1;
        if(it <= row_res[0].data[2]){
            printf("%d---%d,%d\n",row_res[0].data[1],row_res[0].data[2],tid_data);
            row_res[it].data[0] = tag.data[0];
            row_res[it].data[1] = tag.data[1];
            row_res[it].data[2] = tag.data[2];
            row_res[it].data[3] = tag.data[3];
            row_res[it].data[4] = tag.data[4];
        }else{
            ++next;
            add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
        }
    }else{
        Tag5* row_next =  (Tag5*)((char*)index + pitch * ((tid_data + next)%data_size));
        if(row_next[0].data[3] == tid_data){
            int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
            if(new_size >= row_next[0].data[1]){
                printf("%d---%d,%d\n",row_next[0].data[1],row_next[0].data[2],tid_data);
                int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
                row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
                row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
                row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
                row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
                row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
            }else{
                atomicAdd(&row_next[0].data[2], 1);
                ++next;
                add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
            }
        }else if(row_next[0].data[3] == -1 && row_next[0].data[2] > row_next[0].data[1]){
            int old = atomicAdd(&row_next[0].data[3],tid_data+1);
            if(old != -1)
                atomicSub(&row_next[0].data[3],tid_data+1);
            else{
                int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
                if(new_size >= row_next[0].data[1]){
                    printf("%d---%d,%d\n",row_next[0].data[1],row_next[0].data[2],tid_data);
                    int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
                    row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
                    row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
                    row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
                    row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
                    row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
                }else{
                    atomicAdd(&row_next[0].data[2], 1);
                    ++next;
                    add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
                }
            }
        }else{
            ++next;
            add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
        }

    }
    return true;
}


__global__ void data_filter(Tag5* index,
                                size_t pitch,
                                const int* data_node,
                                const int* data_degree,
                                const int* data_label,
                                const int* data_adj,
                                const int* query_node,
                                const int* query_degree,
                                const int* query_label,
                                const int* query_adj,
                                int data_size,
                                int query_size){
    int tid_data = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_query = blockIdx.y * blockDim.y + threadIdx.y;
    if(tid_data< data_size && tid_query< query_size){
        if(query_label[tid_query] != data_label[tid_data]) return;
        if(query_degree[tid_query] > data_degree[tid_data]) return;
        int query_len = query_node[tid_query] + query_degree[tid_query];
        int data_len = data_node[tid_data] + data_degree[tid_data];
        int query_loc = query_node[tid_query], data_loc = data_node[tid_data];
//        printf("%d %d\n",tid_data,tid_query);
        while(query_loc < query_len && data_loc < data_len){
//            printf("%d-%d, %d------%d, %d^^^%d\n",tid_data,tid_query,data_label[data_adj[data_loc]],query_label[query_adj[query_loc]],data_loc,query_loc);
            if(query_label[query_adj[query_loc]] == data_label[data_adj[data_loc]]){
                ++query_loc;
                ++data_loc;
            }else if(query_label[query_adj[query_loc]] < data_label[data_adj[data_loc]]){
                break;
            }else{
                ++data_loc;
            }
        }
        if(query_loc == query_len){
//            printf("-----%d %d\n",tid_data,tid_query);
            Tag5* row_res = (Tag5*)((char*)index + pitch * tid_data);
            Tag5 tag = {0,tid_data,0,0,tid_query};
            add_tag(tag,index,row_res,tid_data,data_size,pitch,0);
        }
    }
}


//    test
__host__ void print_h_index(Tag5* h_index,int data_size){
    cout << "=========" << endl;
    for(int i = 0; i<data_size; ++i){
        cout<<i<<": ";
        for(int j = 0 ; j<h_index[i*N_size].data[0]; ++j){
            cout<<h_index[i*N_size + j].data[0]<<" "
                <<h_index[i*N_size + j].data[1]<<" "
                <<h_index[i*N_size + j].data[2]<<" "
                <<h_index[i*N_size + j].data[3]<<" "
                <<h_index[i*N_size + j].data[4]<<" / ";
        }
        cout<<endl;
    }
}

__global__ void d_print(Tag5* index,size_t pitch,int size,int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        for(int i = 0 ; i< N ;++i){
            printf("%d: %d %d %d %d %d\n",tid,row[i].data[0],row[i].data[1],row[i].data[2],row[i].data[3],row[i].data[4]);
        }

    }
}

__global__ void add_one(Tag5* index,size_t pitch,Tag5 tag, int loc,int data_v_num){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if( tid == 11 || tid == 10 || tid == 0 || tid== 1){
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        add_tag(tag,index,row,tid,data_v_num,pitch,0);
    }
}

__global__ void init_edge(Tag5* index,size_t pitch,
                          const int* query_edge_pair,
                          const int* data_label,
                          const int* data_node,
                          const int* data_degree,
                          const int* data_adj,
                          const int* single_group_name,
                          int data_v_num, int edge_num){
    int tid_data = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_query = blockIdx.y * blockDim.y + threadIdx.y;
    if(tid_data< data_v_num && tid_query< edge_num){
        int node_first = query_edge_pair[2* tid_query];
        int node_second = query_edge_pair[2* tid_query + 1];
        bool flag = false;
        Tag5 *row = (Tag5*)((char*)index + pitch * tid_data);
        for(int i = 1 ; i <= row[0].data[2]; ++i){
            if(row[i].data[0] == 0 &&row[i].data[4] == node_first){
                flag = true;
                break;
            }
        }
        if(flag){
            for(int node = data_node[tid_data]; node <data_node[tid_data] + data_degree[tid_data]; ++node){
                int adj_node = data_adj[node];
                Tag5 *row_adj = (Tag5*)((char*)index + pitch * adj_node);
                for(int i = 1 ; i<= row[0].data[2]; ++i){
                    if(row_adj[i].data[0] == 0 && row_adj[i].data[4] == node_second){
                        Tag5 tag_first = {single_group_name[tid_query],tid_data,i-1,adj_node,node_first};
                        Tag5 tag_second = {single_group_name[tid_query],tid_data,i-1,tid_data,node_second};
                        add_tag(tag_first,index,row,tid_data,data_v_num,pitch,0);
                        add_tag(tag_second,index,row_adj,tid_data,data_v_num,pitch,0);
                        break;
                    }
                }
            }
        }
    }
}