#include "hip/hip_runtime.h"
#include "alg.cuh"
#include "iostream"
__device__ bool add_tag(Tag5 tag,Tag5* index,Tag5* row_res,int tid_data,int data_size, size_t pitch,int next){
    if(next == 0){
        int it = atomicAdd(&row_res[0].data[1], 1) + 1;
        if(it <= row_res[0].data[2]){
            printf("insert self %d---%d,%d\n",row_res[0].data[1],row_res[0].data[2],tid_data);
            row_res[it].data[0] = tag.data[0];
            row_res[it].data[1] = tag.data[1];
            row_res[it].data[2] = tag.data[2];
            row_res[it].data[3] = tag.data[3];
            row_res[it].data[4] = tag.data[4];
            return true;
        }else{
            return false;
        }
    }else{
        printf("try insert %d -to- >%d\n",tid_data,(tid_data + next)%data_size);
        Tag5* row_next =  (Tag5*)((char*)index + pitch * ((tid_data + next)%data_size));
        if(row_next[0].data[3] == tid_data){
            printf("%d == data[3]\n",tid_data);
            int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
            if(new_size >= row_next[0].data[1]){
                printf("insert others %d---%d,%d--to--> %d\n",row_next[0].data[1],row_next[0].data[2],tid_data,(tid_data + next)%data_size);
                int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
                row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
                row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
                row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
                row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
                row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
                return true;
            }else{
                printf("others failed %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
                atomicAdd(&row_next[0].data[2], 1);
                return false;
            }
        }else if(row_next[0].data[3] == -1 && row_next[0].data[2] > row_next[0].data[1]){
            printf("%d == -1 >data[1]\n",tid_data);
            int old = atomicAdd(&row_next[0].data[3],tid_data+1);
            if(old != -1){
                printf("%d --> %d update failed\n",tid_data,(tid_data + next)%data_size);
                atomicSub(&row_next[0].data[3],tid_data+1);
                return false;
            }
            else{
                int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
                if(new_size >= row_next[0].data[1]){
                    printf("%d --> %d update succeed and insert\n",tid_data,(tid_data + next)%data_size);
                    printf("%d---%d,%d\n",row_next[0].data[1],row_next[0].data[2],tid_data);
                    int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
                    row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
                    row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
                    row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
                    row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
                    row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
                    return true;
                }else{
                    printf("others failed but in %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
                    atomicAdd(&row_next[0].data[2], 1);
                    return false;
                }
            }
        }else{
            printf("%d shit!\n",tid_data);
            printf("over and next %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
            return false;
        }
    }
}


//__device__ bool add_tag_old(Tag5 tag,Tag5* index,Tag5* row_res,int tid_data,int data_size, size_t pitch,int next){
//    if(next==0){
//        int it = atomicAdd(&row_res[0].data[1], 1) + 1;
//        if(it <= row_res[0].data[2]){
//            printf("insert self %d---%d,%d\n",row_res[0].data[1],row_res[0].data[2],tid_data);
//            row_res[it].data[0] = tag.data[0];
//            row_res[it].data[1] = tag.data[1];
//            row_res[it].data[2] = tag.data[2];
//            row_res[it].data[3] = tag.data[3];
//            row_res[it].data[4] = tag.data[4];
//        }else{
//            printf("need insert others %d\n",tid_data);
//            ++next;
//            add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//        }
//    }else{
//        printf("try insert %d -to- >%d\n",tid_data,(tid_data + next)%data_size);
//        Tag5* row_next =  (Tag5*)((char*)index + pitch * ((tid_data + next)%data_size));
//        printf("%d --> %d : %d , %d ,%d ,%d\n",tid_data,(tid_data + next)%data_size,row_next[0].data[1],row_next[0].data[2],row_next[0].data[3],row_next[0].data[4]);
//        if(row_next[0].data[3] == tid_data){
//            printf("%d == data[3]\n",tid_data);
//            int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
//            if(new_size >= row_next[0].data[1]){
//                printf("insert others %d---%d,%d--to--> %d\n",row_next[0].data[1],row_next[0].data[2],tid_data,(tid_data + next)%data_size);
//                int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
//                row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
//                row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
//                row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
//                row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
//                row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
//            }else{
//                printf("others failed %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
//                atomicAdd(&row_next[0].data[2], 1);
//                ++next;
//                add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//            }
//        }else if(row_next[0].data[3] == -1 && row_next[0].data[2] > row_next[0].data[1]){
//            printf("%d == -1 >data[1]\n",tid_data);
//            int old = atomicAdd(&row_next[0].data[3],tid_data+1);
//            if(old != -1){
//                printf("%d --> %d update failed\n",tid_data,(tid_data + next)%data_size);
//                atomicSub(&row_next[0].data[3],tid_data+1);
//                ++next;
//                add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//            }
//            else{
//
//                int new_size = atomicSub(&row_next[0].data[2], 1) - 1;
//                if(new_size >= row_next[0].data[1]){
//                    printf("%d --> %d update succeed and insert\n",tid_data,(tid_data + next)%data_size);
//                    printf("%d---%d,%d\n",row_next[0].data[1],row_next[0].data[2],tid_data);
//                    int old_loc_o = atomicAdd(&row_next[0].data[4], 1);
//                    row_next[N_size - old_loc_o -1].data[0] = tag.data[0];
//                    row_next[N_size - old_loc_o -1].data[1] = tag.data[1];
//                    row_next[N_size - old_loc_o -1].data[2] = tag.data[2];
//                    row_next[N_size - old_loc_o -1].data[3] = tag.data[3];
//                    row_next[N_size - old_loc_o -1].data[4] = tag.data[4];
//                }else{
//                    printf("others failed but in %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
//                    atomicAdd(&row_next[0].data[2], 1);
//                    ++next;
//                    add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//                }
//            }
//        }else{
//            printf("%d shit!\n",tid_data);
//            printf("over and next %d --to--> %d\n",tid_data,(tid_data + next)%data_size);
//            ++next;
//            add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
//        }
//
//    }
//    return true;
//}


__global__ void data_filter(Tag5* index,
                                size_t pitch,
                                const int* data_node,
                                const int* data_degree,
                                const int* data_label,
                                const int* data_adj,
                                const int* query_node,
                                const int* query_degree,
                                const int* query_label,
                                const int* query_adj,
                                int data_size,
                                int query_size){
    int tid_data = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_query = blockIdx.y * blockDim.y + threadIdx.y;
    if(tid_data< data_size && tid_query< query_size){
        if(query_label[tid_query] != data_label[tid_data]) return;
        if(query_degree[tid_query] > data_degree[tid_data]) return;
        int query_len = query_node[tid_query] + query_degree[tid_query];
        int data_len = data_node[tid_data] + data_degree[tid_data];
        int query_loc = query_node[tid_query], data_loc = data_node[tid_data];
//        printf("%d %d\n",tid_data,tid_query);
        while(query_loc < query_len && data_loc < data_len){
//            printf("%d-%d, %d------%d, %d^^^%d\n",tid_data,tid_query,data_label[data_adj[data_loc]],query_label[query_adj[query_loc]],data_loc,query_loc);
            if(query_label[query_adj[query_loc]] == data_label[data_adj[data_loc]]){
                ++query_loc;
                ++data_loc;
            }else if(query_label[query_adj[query_loc]] < data_label[data_adj[data_loc]]){
                break;
            }else{
                ++data_loc;
            }
        }
        if(query_loc == query_len){
//            printf("-----%d %d\n",tid_data,tid_query);
            Tag5* row_res = (Tag5*)((char*)index + pitch * tid_data);
            Tag5 tag = {0,tid_data,0,0,tid_query};
            int next = 0;
            bool flag = false;
            while (!flag && next<data_size){
                flag = add_tag(tag,index,row_res,tid_data,data_size,pitch,next);
                ++next;
            }

        }
    }
}


//    test
__host__ void print_h_index(Tag5* h_index,int data_size){
    cout << "=========" << endl;
    for(int i = 0; i<data_size; ++i){
        cout<<i<<": ";
        for(int j = 0 ; j<h_index[i*N_size].data[0]; ++j){
            cout<<h_index[i*N_size + j].data[0]<<" "
                <<h_index[i*N_size + j].data[1]<<" "
                <<h_index[i*N_size + j].data[2]<<" "
                <<h_index[i*N_size + j].data[3]<<" "
                <<h_index[i*N_size + j].data[4]<<" / ";
        }
        cout<<endl;
    }
}

__global__ void d_print(Tag5* index,size_t pitch,int size,int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        for(int i = 0 ; i< N ;++i){
            printf("%d: %d %d %d %d %d\n",tid,row[i].data[0],row[i].data[1],row[i].data[2],row[i].data[3],row[i].data[4]);
        }

    }
}

__global__ void add_one(Tag5* index,size_t pitch,Tag5 tag, int loc,int data_v_num){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if( tid == loc ){
        printf("tid --> %d\n",tid);
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        int next = 0;
        bool flag = false;
        while (!flag && next<data_v_num){
            flag = add_tag(tag,index,row,tid,data_v_num,pitch,next);
            ++next;
        }
    }
}

__global__ void init_edge(Tag5* index,size_t pitch,
                          const int* query_edge_pair,
                          const int* data_label,
                          const int* data_node,
                          const int* data_degree,
                          const int* data_adj,
                          const int* single_group_name,
                          int data_v_num, int edge_num){
    int tid_data = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_query = blockIdx.y * blockDim.y + threadIdx.y;
    if(tid_data< data_v_num && tid_query< edge_num){
        int node_first = query_edge_pair[2* tid_query];
        int node_second = query_edge_pair[2* tid_query + 1];
        bool flag = false;
        Tag5 *row = (Tag5*)((char*)index + pitch * tid_data);
        for(int i = 1 ; i <= row[0].data[1]; ++i){
            if(row[i].data[0] == 0 && row[i].data[4] == node_first){
                flag = true;
                break;
            }
        }
        if(flag){
//            printf("%d--%d == %d %d\n",tid_data,tid_query, node_first, node_second);
            for(int node = data_node[tid_data]; node <data_node[tid_data] + data_degree[tid_data]; ++node){
//                printf("%d, %d ---%d\n",tid_data,node,data_node[tid_data] + data_degree[tid_data]);
                int adj_node = data_adj[node];
//                printf("%d--%d == %d %d ---%d\n",tid_data,tid_query, node_first, node_second ,adj_node);
                Tag5 *row_adj = (Tag5*)((char*)index + pitch * adj_node);
                for(int i = 1 ; i<= row_adj[0].data[1]; ++i){
                    if(row_adj[i].data[0] == 0 && row_adj[i].data[4] == node_second){
                        Tag5 tag_first = {single_group_name[tid_query],tid_data,i-1,adj_node,node_first};
                        Tag5 tag_second = {single_group_name[tid_query],tid_data,i-1,tid_data,node_second};
                        printf("add tag %d, %d\n",tid_data,adj_node);
                        int next = 0;
                        bool ff = false;
                        while(!ff && next<data_v_num){
                            ff = add_tag(tag_first,index,row,tid_data,data_v_num,pitch,next);
                            ++next;
                        }
                        next = 0;
                        ff = false;
                        while(!ff && next<data_v_num){
                            ff = add_tag(tag_second,index,row_adj,adj_node,data_v_num,pitch,next);
                            ++next;
                        }
                        break;
                    }
                }
            }
        }
    }
}

__device__ int find_next(int cur_tid,int ori_tid,Tag5* index,size_t pitch,int data_v_num){
    cur_tid = (cur_tid+1)%data_v_num;
    Tag5 *row = (Tag5*)((char*)index + pitch * cur_tid);
    while (row[0].data[3] != ori_tid){
        printf("%d find %d\n",ori_tid,cur_tid);
        cur_tid = (cur_tid+1)%data_v_num;
        row = (Tag5*)((char*)index + pitch * cur_tid);
    }
    printf("%d ### %d\n", ori_tid,cur_tid);
    return cur_tid;
};

__global__ void joint(Tag5* index,size_t pitch,Tag4 info,
                      const int* del_edge,
                      const int* node_set,
                      const int* degree_set,
                      const int* adj_set,
                      int data_v_num,int query_v_num,int del_edge_size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 10 || tid == 11 || tid == 0){
        Tag2 first_set[N_size/2];
        Tag2 second_set[N_size/2];
        int first_count = 0, second_count = 0;
        Tag5 *row = (Tag5*)((char*)index + pitch * tid);
        for (int i = 1 ; i <= row[0].data[2]; ++i){
            printf("%d: %d %d  info %d %d %d\n", tid, row[i].data[0],row[i].data[4],info.data[0],info.data[1],info.data[2]);
            if(row[i].data[0] == info.data[0] && row[i].data[4] == info.data[2]){
                printf("%d--%d insert first\n",tid,i);
                first_set[first_count++] = {tid,i};
                break;
            }
            if(row[i].data[0] == info.data[1] && row[i].data[4] == info.data[2]){
                printf("%d--%d insert second\n",tid,i);
                second_set[second_count++] = {tid,i};
                break;
            }
        }
        if(row[0].data[1] > row[0].data[2]){
            printf("%d find others\n",tid);
            int cur_count = row[0].data[2];
            int sum_count = row[0].data[1];
            int cur_tid = tid;
            while (cur_count < sum_count){
                int next_tid = find_next(cur_tid,tid,index,pitch,data_v_num);
                printf("%d --next--> %d\n",tid ,next_tid);
                Tag5 *row_next = (Tag5*)((char*)index + pitch * next_tid);
                printf("%d %d %d %d %d\n",row_next[0].data[0],row_next[0].data[1],row_next[0].data[2],row_next[0].data[3],row_next[0].data[4]);
                cur_count += row_next[0].data[4];
                for (int i = 0 ; i < row_next[0].data[4]; ++i){
                    printf("others: %d: %d %d  info %d %d %d\n", tid, row_next[i].data[0],row_next[0].data[4],info.data[0],info.data[1],info.data[2]);
                    if(row_next[N_size- i -1].data[0] == info.data[0] && row_next[N_size- i -1].data[4] == info.data[2]){
                        first_set[first_count++] = {next_tid,N_size- i -1};
                        break;
                    }
                    if(row_next[N_size- i -1].data[0] == info.data[1] && row_next[N_size- i -1].data[4] == info.data[2]){
                        second_set[second_count++] = {next_tid,N_size- i -1};
                        break;
                    }
                }
            }
        }
        for(int i = 0 ;i < second_count; ++i){
            printf("%d %d,%d\n",tid ,second_set[i].data[0],second_set[i].data[1]);
        }

        int new_serial = 0;
        for(int i = 0; i < first_count; ++i){
            Tag5 *first_row = (Tag5*)((char*)index + pitch * first_set[i].data[0]);
            Tag5 first = first_row[first_set[i].data[1]];
            for(int j = 0 ; j < second_count; ++j){
                Tag5 *second_row = (Tag5*)((char*)index + pitch * second_set[j].data[0]);
                Tag5 second = second_row[second_set[j].data[1]];

                int table[MAX_query_Size] = {-1};
                int exist_table[MAX_query_Size];
                int exist_count = 0;
                for(int t = 0; t < query_v_num; ++t){
                    table[t] = -1;
                }

                int group,root,serial,next,match;
                group = first.data[0];
                root = first.data[1];
                serial = first.data[2];
                next = first.data[3];
                match = first.data[4];
                table[match] = tid;
                while(next != tid){
                    Tag5 *row_next = (Tag5*)((char*)index + pitch * next);
                    for(int loc = 1; loc <= row_next[0].data[2]; ++loc){
                        if(row_next[loc].data[0] == group && row_next[loc].data[1] == root && row_next[loc].data[2] == serial){
                            table[row_next[loc].data[4]] = next;
                            exist_table[exist_count++] = next;
                            next = row_next[loc].data[3];
                            break;
                        }
                    }
                }
                group = second.data[0];
                root = second.data[1];
                serial = second.data[2];
                next = second.data[3];

                //unique check
                bool flag_unique_check = true;

                while(next != tid){
                    Tag5 *row_next = (Tag5*)((char*)index + pitch * next);
                    for(int loc = 1; loc <= row_next[0].data[2]; ++loc){
                        if(row_next[loc].data[0] == group && row_next[loc].data[1] == root && row_next[loc].data[2] == serial){
                            for(int check = 0; check < exist_count; ++check){
                                if(next == exist_table[check]){
                                    flag_unique_check = false;
                                    break;
                                }
                            }
                            table[row_next[loc].data[4]] = next;
                            next = row_next[loc].data[3];
                            break;
                        }
                    }
                }
                //single edge check
                bool flag_single_edge_check = false;
                if(del_edge_size> 0 && flag_unique_check){
                    for(int d = 0 ; d < del_edge_size; d = d + 2){
                        int node_a = table[del_edge[d]];
                        int node_b = table[del_edge[d+1]];
                        bool f_temp = false;
                        for(int check = node_set[node_a]; check < node_set[node_a] + degree_set[node_a]; ++check ){
                            if(adj_set[check] == node_b){
                                f_temp = true;
                                break;
                            }
                        }
                        if(f_temp){
                           flag_single_edge_check = true;
                            break;
                        }
                    }
                }
                //init and add new tag
                int slow = info.data[2];
                int fast = (info.data[2]+1) % query_v_num;
                while(fast != info.data[2]){
                    if(table[fast] != -1){
                        Tag5 *row_res = (Tag5*)((char*)index + pitch * table[slow]);
                        Tag5 new_tag = {info.data[3],tid,new_serial++,table[fast],slow};
                        int next_row = 0;
                        bool flag = false;
                        while (!flag && next_row < data_v_num){
                            flag = add_tag(new_tag,index,row_res,table[slow],data_v_num,pitch,next_row);
                            ++next_row;
                        }
                        slow = fast;
                    }
                    fast = (fast+1)%query_v_num;
                }
                Tag5 *row_res = (Tag5*)((char*)index + pitch * table[slow]);
                Tag5 new_tag = {info.data[3],tid,new_serial,table[fast],slow};
                int next_row = 0;
                bool flag = false;
                while (!flag && next_row < data_v_num){
                    flag = add_tag(new_tag,index,row_res,table[slow],data_v_num,pitch,next_row);
                    ++next_row;
                }
            }
        }
    }

}