#include <iostream>
#include "vector"
#include "Graph.cuh"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "alg.cuh"
#include "unordered_set"
using namespace std;


vector<vector<int>> midNode;
vector<vector<Tag4>> joint_group;
vector<int> single_pair;
unordered_map<int,unordered_set<int>> group_name_map;
vector<int> single_pair_name;
int cutStep;

int main() {
//    int device = 0;  // 要查询的设备索引
//    int value;
//
//    hipDeviceProp_t props;
//    hipGetDeviceProperties(&props, device);
//    hipDeviceGetAttribute(&value, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
//
//    std::cout << "Max shared memory per block: " << value << " bytes" << std::endl;

    string query_path = "../test/query";
    string data_path = "../test/data";
    auto *query = new Graph();
    auto *data = new Graph();
    query->ReadInFile(query_path,1);
    data->ReadInFile(data_path,0);
    query->except_ring();
    query->calcLevelId();
    int name = 1;
    group_name_map[name].insert(query->originalId.begin(), query->originalId.end());
    query->division(0,name);
    joint_group.resize(cutStep+1);

    thrust::device_vector<int> dev_data_node(data->q_h_node);
    thrust::device_vector<int> dev_data_label(data->q_h_label);
    thrust::device_vector<int> dev_data_adj(data->q_h_adj);
    thrust::device_vector<int> dev_data_degree(data->q_h_degree);

    thrust::device_vector<int> dev_query_node(query->q_h_node);
    thrust::device_vector<int> dev_query_label(query->q_h_label);
    thrust::device_vector<int> dev_query_adj(query->q_h_adj);
    thrust::device_vector<int> dev_query_degree(query->q_h_degree);

    thrust::device_vector<int> dev_single_pair_name(single_pair_name);
    thrust::device_vector<int> dev_single_pair(single_pair);

    //index
    Tag5 *d_index, *h_index;
    size_t pitch;
    h_index = new Tag5[N_size * data->v_num];
    memset(h_index,-1,N_size * data->v_num*sizeof(Tag5));
    for (int i = 0; i < N_size * data->v_num; i = i + N_size) {
        h_index[i] = {N_size,0, N_size-1, -1, 0};
    }
    hipMallocPitch(&d_index, &pitch, N_size * sizeof(Tag5), data->v_num);
    hipMemcpy2D(d_index, pitch, h_index, N_size * sizeof(Tag5), N_size * sizeof(Tag5), data->v_num,hipMemcpyHostToDevice);


    dim3 grid_2D((data->v_num/32)+1,(query->v_num/32)+1);
    dim3 block_2D(32,32);
    //filter
    data_filter<<<grid_2D,block_2D>>>(d_index,pitch,thrust::raw_pointer_cast(dev_data_node.data()),
                                      thrust::raw_pointer_cast(dev_data_degree.data()),
                                      thrust::raw_pointer_cast(dev_data_label.data()),
                                      thrust::raw_pointer_cast(dev_data_adj.data()),
                                      thrust::raw_pointer_cast(dev_query_node.data()),
                                      thrust::raw_pointer_cast(dev_query_degree.data()),
                                      thrust::raw_pointer_cast(dev_query_label.data()),
                                      thrust::raw_pointer_cast(dev_query_adj.data()),data->v_num,query->v_num);

    hipDeviceSynchronize();

//    test
    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
    print_h_index(h_index,data->v_num);



//    Tag5 tag = {999,999,999,999,999};
//    add_one<<<1,13>>>(d_index,pitch,tag,10,data->v_num);
//    add_one<<<1,13>>>(d_index,pitch,tag,11,data->v_num);
//    add_one<<<1,13>>>(d_index,pitch,tag,10,data->v_num);
//    hipDeviceSynchronize();
//    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
//    print_h_index(h_index,data->v_num);
//
//
//    Tag4 info = {0, 999 , 999,999};
//    joint<<<1,13>>>(d_index,pitch,info, nullptr,thrust::raw_pointer_cast(dev_data_node.data()),
//          thrust::raw_pointer_cast(dev_data_degree.data()),
//          thrust::raw_pointer_cast(dev_data_adj.data()),
//          data->v_num,query->v_num);
//
//    hipDeviceSynchronize();

    grid_2D = dim3 ((data->v_num/32)+1 , (single_pair_name.size()/32)+1);
    init_edge<<<grid_2D,block_2D>>>(d_index,pitch,thrust::raw_pointer_cast(dev_single_pair.data()),
                                    thrust::raw_pointer_cast(dev_data_label.data()),
                                    thrust::raw_pointer_cast(dev_data_node.data()),
                                    thrust::raw_pointer_cast(dev_data_degree.data()),
                                    thrust::raw_pointer_cast(dev_data_adj.data()),
                                    thrust::raw_pointer_cast(dev_single_pair_name.data()),data->v_num,(int)single_pair_name.size());

    hipDeviceSynchronize();
    //    test
    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
    print_h_index(h_index,data->v_num);

    //multi joint
    auto del_edge = query->single_edge;
    for(int level = (int) joint_group.size()-1; level>=0 ; --level){
        cout<<"start level" <<level <<endl;
        for(auto const& info : joint_group[level]){
            vector<bool> used(del_edge.size(), true);
            thrust::host_vector<int> h_del_edge;
            for(int del = 0 ; del < del_edge.size(); del = del + 2){
                if(!used[del]) continue;
                if(group_name_map[info.data[3]].count(del_edge[del]) > 0 &&group_name_map[info.data[3]].count(del_edge[del+1]) > 0){
                    h_del_edge.push_back(del_edge[del]);
                    h_del_edge.push_back(del_edge[del+1]);
                    used[del] = false;
                }
            }
            thrust::device_vector<int> d_del_edge = h_del_edge;
            joint<<<data->v_num/BLOCK + 1,BLOCK>>>(d_index,pitch,info,
                                                   thrust::raw_pointer_cast(d_del_edge.data()),
                                                   thrust::raw_pointer_cast(dev_data_node.data()),
                                                   thrust::raw_pointer_cast(dev_data_degree.data()),
                                                   thrust::raw_pointer_cast(dev_data_adj.data()),
                                                   data->v_num,query->v_num,(int)h_del_edge.size());
            hipDeviceSynchronize();
            hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
            print_h_index(h_index,data->v_num);

            int a = 0;

        }
        hipDeviceSynchronize();
    }
    //    test
    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
    print_h_index(h_index,data->v_num);

    thrust::device_vector<int> sum_res(data->v_num);
    sum_count<<<data->v_num/BLOCK + 1,BLOCK>>>(d_index,pitch,1,data->v_num,thrust::raw_pointer_cast(sum_res.data()));
    hipDeviceSynchronize();

    int ret = thrust::reduce(sum_res.begin(), sum_res.end());
    cout<<"count: "<<ret/query->v_num<<endl;

    return 0;
}
