#include <iostream>
#include "vector"
#include "Graph.cuh"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "alg.cuh"
#include "unordered_set"
using namespace std;


vector<vector<int>> midNode;
vector<vector<Tag3>> joint_group;
vector<int> single_pair;
unordered_map<int,vector<int>> group_name_map;
vector<int> single_pair_name;
int cutStep;

int main() {
//    int device = 0;  // 要查询的设备索引
//    int value;
//
//    hipDeviceProp_t props;
//    hipGetDeviceProperties(&props, device);
//    hipDeviceGetAttribute(&value, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
//
//    std::cout << "Max shared memory per block: " << value << " bytes" << std::endl;

    string query_path = "../test/query";
    string data_path = "../test/data";
    auto *query = new Graph();
    auto *data = new Graph();
    query->ReadInFile(query_path,1);
    data->ReadInFile(data_path,0);
    query->except_ring();
    query->calcLevelId();
    int name = 0;
    query->division(0,name);
    joint_group.resize(cutStep+1);

    thrust::device_vector<int> dev_data_node(data->q_h_node);
    thrust::device_vector<int> dev_data_label(data->q_h_label);
    thrust::device_vector<int> dev_data_adj(data->q_h_adj);
    thrust::device_vector<int> dev_data_degree(data->q_h_degree);

    thrust::device_vector<int> dev_query_node(query->q_h_node);
    thrust::device_vector<int> dev_query_label(query->q_h_label);
    thrust::device_vector<int> dev_query_adj(query->q_h_adj);
    thrust::device_vector<int> dev_query_degree(query->q_h_degree);

    thrust::device_vector<int> dev_single_pair_name(single_pair_name);
    thrust::device_vector<int> dev_single_pair(single_pair);

    //index
    Tag5 *d_index, *h_index;
    size_t pitch;
    h_index = new Tag5[N_size * data->v_num];
    memset(h_index,-1,N_size * data->v_num*sizeof(Tag5));
    for (int i = 0; i < N_size * data->v_num; i = i + N_size) {
        h_index[i] = {N_size,0, N_size-1, -1, 0};
    }
    hipMallocPitch(&d_index, &pitch, N_size * sizeof(Tag5), data->v_num);
    hipMemcpy2D(d_index, pitch, h_index, N_size * sizeof(Tag5), N_size * sizeof(Tag5), data->v_num,hipMemcpyHostToDevice);


    dim3 grid_2D((data->v_num/32)+1,(query->v_num/32)+1);
    dim3 block_2D(32,32);
    //filter
    data_filter<<<grid_2D,block_2D>>>(d_index,pitch,thrust::raw_pointer_cast(dev_data_node.data()),
                                      thrust::raw_pointer_cast(dev_data_degree.data()),
                                      thrust::raw_pointer_cast(dev_data_label.data()),
                                      thrust::raw_pointer_cast(dev_data_adj.data()),
                                      thrust::raw_pointer_cast(dev_query_node.data()),
                                      thrust::raw_pointer_cast(dev_query_degree.data()),
                                      thrust::raw_pointer_cast(dev_query_label.data()),
                                      thrust::raw_pointer_cast(dev_query_adj.data()),data->v_num,query->v_num);

    hipDeviceSynchronize();

//    test
    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
    print_h_index(h_index,data->v_num);


    cout<<"==============="<<endl;
    d_print<<<(data->v_num/32)+1,32>>>(d_index,pitch,data->v_num,N_size);
    hipDeviceSynchronize();

    Tag5 tag = {999,999,999,999,999};
    add_one<<<1,13>>>(d_index,pitch,tag,11,data->v_num);
    hipDeviceSynchronize();
    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
    print_h_index(h_index,data->v_num);

    return 0;
//    Tag3 = {, 4 ,};



    grid_2D = dim3 ((data->v_num/32)+1 , (single_pair_name.size()/32)+1);
    init_edge<<<grid_2D,block_2D>>>(d_index,pitch,thrust::raw_pointer_cast(dev_single_pair.data()),
                                    thrust::raw_pointer_cast(dev_data_label.data()),
                                    thrust::raw_pointer_cast(dev_data_node.data()),
                                    thrust::raw_pointer_cast(dev_data_degree.data()),
                                    thrust::raw_pointer_cast(dev_data_adj.data()),
                                    thrust::raw_pointer_cast(dev_single_pair_name.data()),data->v_num,(int)single_pair_name.size());

    hipDeviceSynchronize();
    //    test
    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
    print_h_index(h_index,data->v_num);



//    add_one<<<1,13>>>(d_index,pitch,tag,11,data->v_num);
//    hipDeviceSynchronize();
//    hipMemcpy2D(h_index, N_size * sizeof(Tag5), d_index, pitch, N_size * sizeof(Tag5), data->v_num,hipMemcpyDeviceToHost);
//    print_h_index(h_index,data->v_num);

    return 0;
}
